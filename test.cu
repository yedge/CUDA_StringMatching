
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void test(char* a, int* b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
	int 	dec[7] = {1, 1, 1, 1, 1, 1, 0};
	char 	str[7] = "Hello ";
	
	printf("%s", str);

	int* 	cuda_mem_int;
	char* 	cuda_mem_str;

	hipMalloc((void**)&cuda_mem_str, sizeof(str));
	hipMalloc((void**)&cuda_mem_int, sizeof(dec));

	hipMemcpy(cuda_mem_str, str, sizeof(str), hipMemcpyHostToDevice);
	hipMemcpy(cuda_mem_int, dec, sizeof(dec), hipMemcpyHostToDevice);

	dim3 dimBlock(7);
	dim3 dimGrid(1);

	test<<<dimGrid, dimBlock>>>(cuda_mem_str, cuda_mem_int);

	hipMemcpy(str, cuda_mem_str, sizeof(str), hipMemcpyDeviceToHost);

	hipFree(cuda_mem_str);

	printf("%s\n", str);

	return 1;
}
